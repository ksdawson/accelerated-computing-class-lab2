// FMA Latency Measurement Kernels
//
// This file contains various CUDA kernels to measure the latency of
// fused multiply-add (FMA) operations under different execution patterns:
// - Basic latency measurement
// - Interleaved execution (ILP)
// - Non-interleaved execution (sequential chains)

#include <hip/hip_runtime.h>
#include <iostream>

using data_type = float;

// Inline assembly macro to read GPU cycle counter
#define clock_cycle() \
    ({ \
        unsigned long long ret; \
        asm volatile("mov.u64 %0, %%clock64;" : "=l"(ret)); \
        ret; \
    })

////////////////////////////////////////////////////////////////////////////////
// Basic FMA Latency

__global__ void
fma_latency(data_type *n, unsigned long long *d_start, unsigned long long *d_end) {
    unsigned long long start_time = clock_cycle();
    unsigned long long end_time = clock_cycle();
    data_type x = *n;
    // Memory fence to ensure that the reads are done.
    __threadfence();
    start_time = clock_cycle();

    /// <--- /your code here --->
    x += x * x; // 1 op
    x += x * x;
    x += x * x;
    x += x * x;
    x += x * x; // 5 ops
    x += x * x;
    x += x * x;
    x += x * x;
    x += x * x;
    x += x * x; // 10 ops
    x += x * x;
    x += x * x;
    x += x * x;
    x += x * x;
    x += x * x;
    x += x * x;
    x += x * x;
    x += x * x;
    x += x * x;
    x += x * x; // 20 ops

    end_time = clock_cycle();

    *n = x;
    *d_start = start_time;
    *d_end = end_time;
}

////////////////////////////////////////////////////////////////////////////////
// FMA Latency + Instruction Level Parallelism (Interleaved)

__global__ void fma_latency_interleaved(
    data_type *n,
    unsigned long long *d_start,
    unsigned long long *d_end) {
    unsigned long long start_time = clock_cycle();
    unsigned long long end_time = clock_cycle();

    data_type x = *n;
    data_type y = *n;
    // Memory fence to ensure that the reads are done.
    __threadfence();

    start_time = clock_cycle();

    /// <--- /your code here --->
    x += x * x;
    y += y * y; // 1 op
    x += x * x;
    y += y * y;
    x += x * x;
    y += y * y;
    x += x * x;
    y += y * y;
    x += x * x;
    y += y * y; // 5 ops
    x += x * x;
    y += y * y;
    x += x * x;
    y += y * y;
    x += x * x;
    y += y * y;
    x += x * x;
    y += y * y;
    x += x * x;
    y += y * y; // 10 ops
    x += x * x;
    y += y * y;
    x += x * x;
    y += y * y;
    x += x * x;
    y += y * y;
    x += x * x;
    y += y * y;
    x += x * x;
    y += y * y;
    x += x * x;
    y += y * y;
    x += x * x;
    y += y * y;
    x += x * x;
    y += y * y;
    x += x * x;
    y += y * y;
    x += x * x;
    y += y * y; // 20 ops

    end_time = clock_cycle();

    *n = x + y;
    *d_start = start_time;
    *d_end = end_time;
}

////////////////////////////////////////////////////////////////////////////////
// FMA Latency + Sequential Execution (No Interleaving)

__global__ void fma_latency_no_interleave(
    data_type *n,
    unsigned long long *d_start,
    unsigned long long *d_end) {

    unsigned long long start_time = clock_cycle();
    unsigned long long end_time = clock_cycle();

    data_type x = *n;
    data_type y = *n;
    // Memory fence to ensure that the reads are done.
    __threadfence();

    start_time = clock_cycle();

    /// <--- /your code here --->
    x += x * x; // 1 op
    x += x * x;
    x += x * x;
    x += x * x;
    x += x * x; // 5 ops
    x += x * x;
    x += x * x;
    x += x * x;
    x += x * x;
    x += x * x; // 10 ops
    x += x * x;
    x += x * x;
    x += x * x;
    x += x * x;
    x += x * x;
    x += x * x;
    x += x * x;
    x += x * x;
    x += x * x;
    x += x * x; // 20 ops
    y += y * y; // 1 op
    y += y * y;
    y += y * y;
    y += y * y;
    y += y * y; // 5 ops
    y += y * y;
    y += y * y;
    y += y * y;
    y += y * y;
    y += y * y; // 10 ops
    y += y * y;
    y += y * y;
    y += y * y;
    y += y * y;
    y += y * y;
    y += y * y;
    y += y * y;
    y += y * y;
    y += y * y;
    y += y * y; // 20 ops

    end_time = clock_cycle();

    *n = x + y;
    *d_start = start_time;
    *d_end = end_time;
}

////////////////////////////////////////////////////////////////////////////////
///          YOU DO NOT NEED TO MODIFY THE CODE BELOW HERE.                  ///
////////////////////////////////////////////////////////////////////////////////

// CUDA error checking macro
#define CUDA_CHECK(x) \
    do { \
        hipError_t err = (x); \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error " << static_cast<int>(err) << " (" \
                      << hipGetErrorString(err) << ") at " << __FILE__ << ":" \
                      << __LINE__ << std::endl; \
            std::exit(EXIT_FAILURE); \
        } \
    } while (0)

// Macro to run kernel and print timing results.
#define run_kernel_and_print(kernel, d_n, d_start, d_end) \
    do { \
        unsigned long long h_time_start = 0ull, h_time_end = 0ull; \
        data_type result = 0.0f; \
\
        kernel<<<1, 1>>>(d_n, d_start, d_end); \
        CUDA_CHECK(hipDeviceSynchronize()); \
        CUDA_CHECK(hipMemcpy( \
            &h_time_start, \
            d_start, \
            sizeof(unsigned long long), \
            hipMemcpyDeviceToHost)); \
        CUDA_CHECK(hipMemcpy( \
            &h_time_end, \
            d_end, \
            sizeof(unsigned long long), \
            hipMemcpyDeviceToHost)); \
        CUDA_CHECK(hipMemcpy(&result, d_n, sizeof(data_type), hipMemcpyDeviceToHost)); \
\
        std::cout << "Latency of " << #kernel \
                  << " code snippet = " << (h_time_end - h_time_start) << " cycles" \
                  << std::endl; \
    } while (0)

int main() {
    data_type *d_n = nullptr;
    unsigned long long *d_start = nullptr;
    unsigned long long *d_end = nullptr;

    data_type host_val = 4.0f;
    unsigned long long host_init_time = 0ull;

    CUDA_CHECK(hipMalloc(&d_n, sizeof(data_type)));
    CUDA_CHECK(hipMalloc(&d_start, sizeof(unsigned long long)));
    CUDA_CHECK(hipMalloc(&d_end, sizeof(unsigned long long)));

    CUDA_CHECK(hipMemcpy(d_n, &host_val, sizeof(data_type), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(
        d_start,
        &host_init_time,
        sizeof(unsigned long long),
        hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(
        d_end,
        &host_init_time,
        sizeof(unsigned long long),
        hipMemcpyHostToDevice));

    run_kernel_and_print(fma_latency, d_n, d_start, d_end);
    run_kernel_and_print(fma_latency_interleaved, d_n, d_start, d_end);
    run_kernel_and_print(fma_latency_no_interleave, d_n, d_start, d_end);

    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipFree(d_n));
    CUDA_CHECK(hipFree(d_start));
    CUDA_CHECK(hipFree(d_end));

    return 0;
}